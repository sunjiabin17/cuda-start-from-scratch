#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

__global__ void dynamic_parallelism_check(int size, int depth)
{
    printf(" Depth : %d - tid : %d \n", depth, threadIdx.x);

    if (size == 1)
        return;

    if (threadIdx.x == 0)
    {
        dynamic_parallelism_check<<<1, size / 2>>>(size / 2, depth + 1);
    }
}

int main(int argc, char **argv)
{
    // nvcc -arch=sm_61 -rdc=true -o build/cuda_bin dynamic_parallelism.cu
    dynamic_parallelism_check<<<1, 16>>>(16, 0);
    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}